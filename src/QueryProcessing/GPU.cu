#include "hip/hip_runtime.h"
#include "../../include/QueryProcessing/GPU.hpp"
#include "../../include/Utilities/ErrorHandling.hpp"
#include <iostream>
#include <algorithm>
#include <cstring>

// CUDA kernels

__device__ int strcmp_device(const char* a, const char* b) {
    while (*a && (*a == *b)) {
        a++;
        b++;
    }
    return *(const unsigned char*)a - *(const unsigned char*)b;
}



__global__ void compareStringColumns(
     char** leftColumn, 
     char** rightColumn,
    int leftSize, 
    int rightSize,
    uint8_t* results, 
    int opType) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < leftSize && j < rightSize) {
        uint8_t match = 0;
        
        switch (opType) {
            case 0: // Equals
                match = (strcmp_device(leftColumn[i], rightColumn[j]) == 0) ? 1 : 0;
                break;
            case 1: // Not Equals
                match = (strcmp_device(leftColumn[i], rightColumn[j]) != 0) ? 1 : 0;
                break;
            case 2: // Less Than
                match = (strcmp_device(leftColumn[i], rightColumn[j]) < 0) ? 1 : 0;
                break;
            case 3: // Greater Than
                match = (strcmp_device(leftColumn[i], rightColumn[j]) > 0) ? 1 : 0;
                break;
            case 4: // Less Than or Equals
                match = (strcmp_device(leftColumn[i], rightColumn[j]) <= 0) ? 1 : 0;
                break;
            case 5: // Greater Than or Equals
                match = (strcmp_device(leftColumn[i], rightColumn[j]) >= 0) ? 1 : 0;
                break;
        }
        
        results[i * rightSize + j] = match;
    }
}

__global__ void compareIntColumns(
     int* leftColumn, 
     int* rightColumn,
    int leftSize, 
    int rightSize,
    uint8_t* results, 
    int opType) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < leftSize && j < rightSize) {
        uint8_t match = 0;
        
        switch (opType) {
            case 0: // Equals
                match = (leftColumn[i] == rightColumn[j]) ? 1 : 0;
                break;
            case 1: // Not Equals
                match = (leftColumn[i] != rightColumn[j]) ? 1 : 0;
                break;
            case 2: // Less Than
                match = (leftColumn[i] < rightColumn[j]) ? 1 : 0;
                break;
            case 3: // Greater Than
                match = (leftColumn[i] > rightColumn[j]) ? 1 : 0;
                break;
            case 4: // Less Than or Equals
                match = (leftColumn[i] <= rightColumn[j]) ? 1 : 0;
                break;
            case 5: // Greater Than or Equals
                match = (leftColumn[i] >= rightColumn[j]) ? 1 : 0;
                break;
        }
        
        results[i * rightSize + j] = match;
    }
}

__global__ void compareIntWithConstant(
     int* column, 
    int constant,
    int size, 
    uint8_t* results, 
    int opType) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        uint8_t match = 0;
        
        switch (opType) {
            case 0: // Equals
                match = (column[i] == constant) ? 1 : 0;
                break;
            case 1: // Not Equals
                match = (column[i] != constant) ? 1 : 0;
                break;
            case 2: // Less Than
                match = (column[i] < constant) ? 1 : 0;
                break;
            case 3: // Greater Than
                match = (column[i] > constant) ? 1 : 0;
                break;
            case 4: // Less Than or Equals
                match = (column[i] <= constant) ? 1 : 0;
                break;
            case 5: // Greater Than or Equals
                match = (column[i] >= constant) ? 1 : 0;
                break;
        }
        
        results[i] = match;
    }
}

__global__ void compareStringWithConstant(
    const char** column, 
    char* constant,
    int size, 
    uint8_t* results, 
    int opType) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        uint8_t match = 0;
        
        switch (opType) {
            case 0: // Equals
                match = (strcmp_device(column[i], constant) == 0) ? 1 : 0;
                break;
            case 1: // Not Equals
                match = (strcmp_device(column[i], constant) != 0) ? 1 : 0;
                break;
            case 2: // Less Than
                match = (strcmp_device(column[i], constant) < 0) ? 1 : 0;
                break;
            case 3: // Greater Than
                match = (strcmp_device(column[i], constant) > 0) ? 1 : 0;
                break;
            case 4: // Less Than or Equals
                match = (strcmp_device(column[i], constant) <= 0) ? 1 : 0;
                break;
            case 5: // Greater Than or Equals
                match = (strcmp_device(column[i], constant) >= 0) ? 1 : 0;
                break;
        }
        
        results[i] = match;
    }
}

__global__ void combineResults(
    const uint8_t* results1, 
    const uint8_t* results2,
    uint8_t* output, 
    int size, 
    uint8_t isAnd) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        bool andResult = results1[i] && results2[i];
        bool orResult = results1[i] || results2[i];
        output[i] = (isAnd * andResult) | ((!isAnd) * orResult);
    }
}

GPUManager::GPUManager() {
    // Check if CUDA is available
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess || deviceCount == 0) {
        std::cout << "No CUDA-capable GPU found. Using CPU processing." << std::endl;
        hasGPU_ = false;
    } else {
        std::cout << "GPU acceleration available. Found " << deviceCount << " CUDA device(s)." << std::endl;
        hasGPU_ = true;
    }
}

GPUManager::~GPUManager() {
    // Clean up any GPU resources if needed
}

bool GPUManager::isGPUAvailable() const {
    return hasGPU_;
}

int GPUManager::findColumnIndex(const Table& table, const char* columnName, const char* tableName) {
    const auto& headers = table.getHeaders();
    
    for (size_t i = 0; i < headers.size(); i++) {
        // If table name is specified, check for "tableName.columnName" format
        if (tableName) {
            std::string fullColumnName = std::string(tableName) + "." + std::string(columnName);
            if (headers[i] == fullColumnName || headers[i] == std::string(columnName)) {
                return static_cast<int>(i);
            }
        } 
        // Otherwise check for just the column name
        else if (headers[i] == columnName) {
            return static_cast<int>(i);
        }
    }
    
    return -1; // Column not found
}

std::vector<uint8_t> GPUManager::gpuJoinTables(
    const Table& leftTable, 
    const Table& rightTable,
    const hsql::Expr* conditions) 
{
    if (!hasGPU_) {
        throw std::runtime_error("GPU operations not available");
    }
    
    int leftSize = leftTable.getSize();
    int rightSize = rightTable.getSize();
    int resultSize = leftSize * rightSize;
    
    std::vector<uint8_t> resultVector(resultSize, 0);
    
    // Process each condition and combine results
    if (conditions->type == hsql::kExprOperator) {
        if (conditions->opType == hsql::OperatorType::kOpAnd || conditions->opType == hsql::OperatorType::kOpOr) {
            // Process binary kOpAnd/kOpOr operations
            auto leftResults = processBinaryExpr(leftTable, rightTable, conditions->expr);
            auto rightResults = processBinaryExpr(leftTable, rightTable, conditions->expr2);
            
            // Create device vectors
            uint8_t *d_leftResults, *d_rightResults, *d_output;
            hipMalloc(&d_leftResults, resultSize * sizeof(uint8_t));
            hipMalloc(&d_rightResults, resultSize * sizeof(uint8_t));
            hipMalloc(&d_output, resultSize * sizeof(uint8_t));
            
            // Copy data to device
            hipMemcpy(d_leftResults, leftResults.data(), resultSize * sizeof(uint8_t), hipMemcpyHostToDevice);
            hipMemcpy(d_rightResults, rightResults.data(), resultSize * sizeof(uint8_t), hipMemcpyHostToDevice);
            
            // Set up kernel execution parameters
            int blockSize = 256;
            int numBlocks = (resultSize + blockSize - 1) / blockSize;
            
            // Execute kernel
            uint8_t isAnd = conditions->opType == hsql::OperatorType::kOpAnd ? 1 : 0;
            combineResults<<<numBlocks, blockSize>>>(d_leftResults, d_rightResults, d_output, resultSize, isAnd);
            
            // Copy results back to host
            hipMemcpy(resultVector.data(), d_output, resultSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
            
            // Free device memory
            hipFree(d_leftResults);
            hipFree(d_rightResults);
            hipFree(d_output);
        } 
        else {
            // Process comparison operation
            resultVector = processComparisonExpr(leftTable, rightTable, conditions);
        }
    }
    
    return resultVector;
}

std::shared_ptr<Table> GPUManager::executeJoin(std::shared_ptr<Table> leftTable,
                                             std::shared_ptr<Table> rightTable,
                                             const hsql::Expr* condition) {
    // Get GPU join mask
    auto mask = gpuJoinTables(*leftTable, *rightTable, condition);
    
    // Create result table structure
    auto headers = combineHeaders(*leftTable, *rightTable);
    auto data = mergeJoinResults(*leftTable, *rightTable, mask);
    
    return std::make_shared<Table>(
        leftTable->getName() + "_joined_" + rightTable->getName(),
        headers,
        data
    );
}

std::shared_ptr<Table> GPUManager::applyFilter(const Table& table, 
                                             const std::vector<uint8_t>& mask) 
{
    return std::make_shared<Table>(
        table.getName() + "_filtered",
        table.getHeaders(),
        mergeFilterResults(table, mask)
    );
}

std::vector<std::vector<std::string>> GPUManager::mergeFilterResults(
    const Table& table,
    const std::vector<uint8_t>& mask) const 
{
    std::vector<std::vector<std::string>> result;
    
    #pragma omp parallel for
    for (size_t i = 0; i < mask.size(); ++i) {
        if (mask[i]) {
            #pragma omp critical
            result.push_back(table.getRow(i));
        }
    }
    
    return result;
}

std::vector<std::string> GPUManager::combineHeaders(const Table& left,
                                                  const Table& right) const {
    std::vector<std::string> headers;
    
    // Add left headers with alias
    for (const auto& h : left.getHeaders()) {
        headers.push_back(left.getAlias().empty() ? h : left.getAlias() + "." + h);
    }
    
    // Add right headers with alias
    for (const auto& h : right.getHeaders()) {
        headers.push_back(right.getAlias().empty() ? h : right.getAlias() + "." + h);
    }
    
    return headers;
}

std::vector<std::vector<std::string>> GPUManager::mergeJoinResults(
    const Table& left,
    const Table& right,
    const std::vector<uint8_t>& mask) const {
    
    std::vector<std::vector<std::string>> result;
    const size_t rightSize = right.getSize();
    
    #pragma omp parallel for
    for (size_t idx = 0; idx < mask.size(); ++idx) {
        if (mask[idx]) {
            // Calculate row indices
            const size_t leftIdx = idx / rightSize;
            const size_t rightIdx = idx % rightSize;
            
            // Combine rows
            auto combined = left.getRow(leftIdx);
            const auto& rightRow = right.getRow(rightIdx);
            combined.insert(combined.end(), rightRow.begin(), rightRow.end());
            
            #pragma omp critical
            result.push_back(std::move(combined));
        }
    }
    
    return result;
}

std::vector<uint8_t> GPUManager::gpuFilterTable(
    const Table& table,
    const hsql::Expr* conditions) 
{
    if (!hasGPU_) {
        throw std::runtime_error("GPU operations not available");
    }
    
    int tableSize = table.getSize();
    std::vector<uint8_t> resultVector(tableSize, 0);
    
    // Process the conditions (simplified for the example)
    if (conditions->type == hsql::kExprOperator) {
        // Handle comparison with constant
        if (conditions->expr->type == hsql::kExprColumnRef && 
            (conditions->expr2->type == hsql::kExprLiteralInt || 
             conditions->expr2->type == hsql::kExprLiteralString)) {
            
            const char* columnName = conditions->expr->name;
            int columnIndex = findColumnIndex(table, columnName, conditions->expr->table);
            
            if (columnIndex == -1) {
                throw std::runtime_error("Column not found: " + std::string(columnName));
            }
            
            uint8_t* d_results;
            hipMalloc(&d_results, tableSize * sizeof(uint8_t));
            
            int blockSize = 256;
            int numBlocks = (tableSize + blockSize - 1) / blockSize;
            
            // Convert operator type to our internal representation
            int opType;
            switch (conditions->opType) {
                case hsql::OperatorType::kOpEquals: opType = 0; break;
                case hsql::OperatorType::kOpNotEquals: opType = 1; break;
                case hsql::OperatorType::kOpLess: opType = 2; break;
                case hsql::OperatorType::kOpGreater: opType = 3; break;
                case hsql::OperatorType::kOpLessEq: opType = 4; break;
                case hsql::OperatorType::kOpGreaterEq: opType = 5; break;
                default: throw std::runtime_error("Unsupported operator type");
            }
            
            // For integer comparison
            if (conditions->expr2->type == hsql::kExprLiteralInt) {
                int constant = conditions->expr2->ival;
                
                // Prepare column data for GPU
                std::vector<int> columnData(tableSize);
                const auto& data = table.getData();
                for (int i = 0; i < tableSize; i++) {
                    columnData[i] = std::stoi(data[i][columnIndex]);
                }
                
                int* d_column;
                hipMalloc(&d_column, tableSize * sizeof(int));
                hipMemcpy(d_column, columnData.data(), tableSize * sizeof(int), hipMemcpyHostToDevice);
                
                compareIntWithConstant<<<numBlocks, blockSize>>>(d_column, constant, tableSize, d_results, opType);
                hipFree(d_column);
            }
            // For string comparison
            else if (conditions->expr2->type == hsql::kExprLiteralString) {
                const char* constant = conditions->expr2->name;
                
                // Prepare column data for GPU (simplified - in reality, this would be more complex for strings)
                std::vector<const char*> columnData(tableSize);
                const auto& data = table.getData();
                for (int i = 0; i < tableSize; i++) {
                    columnData[i] = data[i][columnIndex].c_str();
                }
                
                // Note: This is a simplified approach. In a real implementation,
                // handling strings in CUDA would be more complex.
                const char** d_column;
                char* d_constant;
                hipMalloc(&d_column, tableSize * sizeof(char*));
                hipMalloc(&d_constant, strlen(constant) + 1);
                
                hipMemcpy(d_column, columnData.data(), tableSize * sizeof(char*), hipMemcpyHostToDevice);
                hipMemcpy(d_constant, constant, strlen(constant) + 1, hipMemcpyHostToDevice);
                
                compareStringWithConstant<<<numBlocks, blockSize>>>(d_column, d_constant, tableSize, d_results, opType);
                
                hipFree(d_column);
                hipFree(d_constant);
            }
            
            // Copy results back
            hipMemcpy(resultVector.data(), d_results, tableSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
            hipFree(d_results);
        }
    }
    
    return resultVector;
}

// Helper function to check if a string can be parsed as an integer
bool isInteger(const std::string& str) {
    if (str.empty()) return false;
    
    size_t start = 0;
    if (str[0] == '-' || str[0] == '+') {
        if (str.size() == 1) return false;
        start = 1;
    }
    
    for (size_t i = start; i < str.size(); i++) {
        if (!std::isdigit(str[i])) return false;
    }
    
    return true;
}

// Main function with both string and integer handling
std::vector<uint8_t> GPUManager::processComparisonExpr(
    const Table& leftTable, 
    const Table& rightTable,
    const hsql::Expr* expr) 
{
    if (!expr || expr->type != hsql::kExprOperator) {
        throw std::runtime_error("Expected comparison expression");
    }
    
    int leftSize = leftTable.getSize();
    int rightSize = rightTable.getSize();
    int resultSize = leftSize * rightSize;
    
    std::vector<uint8_t> resultVector(resultSize, 0);
    
    // Column-column comparison
    if (expr->expr->type == hsql::kExprColumnRef && expr->expr2->type == hsql::kExprColumnRef) {
        const char* leftColName = expr->expr->name;
        const char* rightColName = expr->expr2->name;
        
        int leftColIndex = findColumnIndex(leftTable, leftColName, expr->expr->table);
        int rightColIndex = findColumnIndex(rightTable, rightColName, expr->expr2->table);
        
        if (leftColIndex == -1 || rightColIndex == -1) {
            throw std::runtime_error("Column not found in comparison");
        }
        
        // Determine the operator type
        int opType;
        switch (expr->opType) {
            case hsql::OperatorType::kOpEquals: opType = 0; break;
            case hsql::OperatorType::kOpNotEquals: opType = 1; break;
            case hsql::OperatorType::kOpLess: opType = 2; break;
            case hsql::OperatorType::kOpGreater: opType = 3; break;
            case hsql::OperatorType::kOpLessEq: opType = 4; break;
            case hsql::OperatorType::kOpGreaterEq: opType = 5; break;
            default: throw std::runtime_error("Unsupported operator type");
        }
        
        const auto& leftData = leftTable.getData();
        const auto& rightData = rightTable.getData();
        
        // Check if we're dealing with integer or string columns
        bool isIntegerComparison = false;
        if (!leftData.empty() && !rightData.empty()) {
            // Sample the first row of each table to determine type
            isIntegerComparison = isInteger(leftData[0][leftColIndex]) && 
                                 isInteger(rightData[0][rightColIndex]);
        }
        
        // Set up grid and block dimensions for 2D execution
        dim3 blockDim(16, 16);
        dim3 gridDim(
            (leftSize + blockDim.x - 1) / blockDim.x,
            (rightSize + blockDim.y - 1) / blockDim.y
        );
        
        if (isIntegerComparison) {
            // Handle integer columns
            std::vector<int> leftColData(leftSize);
            std::vector<int> rightColData(rightSize);
            
            // Prepare column data
            for (int i = 0; i < leftSize; i++) {
                leftColData[i] = std::stoi(leftData[i][leftColIndex]);
            }
            
            for (int i = 0; i < rightSize; i++) {
                rightColData[i] = std::stoi(rightData[i][rightColIndex]);
            }
            
            // Allocate device memory
            int *d_leftCol, *d_rightCol;
            uint8_t *d_results;
            
            hipMalloc(&d_leftCol, leftSize * sizeof(int));
            hipMalloc(&d_rightCol, rightSize * sizeof(int));
            hipMalloc(&d_results, resultSize * sizeof(uint8_t));
            
            // Copy data to device
            hipMemcpy(d_leftCol, leftColData.data(), leftSize * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_rightCol, rightColData.data(), rightSize * sizeof(int), hipMemcpyHostToDevice);
            
            // Launch integer comparison kernel
            compareIntColumns<<<gridDim, blockDim>>>(
                d_leftCol, d_rightCol, leftSize, rightSize, d_results, opType);
            
            // Copy results back to host
            hipMemcpy(resultVector.data(), d_results, resultSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
            
            // Free device memory
            hipFree(d_leftCol);
            hipFree(d_rightCol);
            hipFree(d_results);
        } else {
            // Handle string columns
            std::vector<std::string> leftColData(leftSize);
            std::vector<std::string> rightColData(rightSize);
            
            // Prepare column data
            for (int i = 0; i < leftSize; i++) {
                leftColData[i] = leftData[i][leftColIndex];
            }
            
            for (int i = 0; i < rightSize; i++) {
                rightColData[i] = rightData[i][rightColIndex];
            }
            
            // Create array of C-style strings on device
            char** h_leftStrings = new char*[leftSize];
            char** h_rightStrings = new char*[rightSize];
            
            // Allocate memory for each string on device
            for (int i = 0; i < leftSize; i++) {
                hipMalloc(&h_leftStrings[i], leftColData[i].size() + 1);
                hipMemcpy(h_leftStrings[i], leftColData[i].c_str(), 
                          leftColData[i].size() + 1, hipMemcpyHostToDevice);
            }
            
            for (int i = 0; i < rightSize; i++) {
                hipMalloc(&h_rightStrings[i], rightColData[i].size() + 1);
                hipMemcpy(h_rightStrings[i], rightColData[i].c_str(), 
                          rightColData[i].size() + 1, hipMemcpyHostToDevice);
            }
            
            // Copy arrays of pointers to device
            char** d_leftStrings, **d_rightStrings;
            uint8_t* d_results;
            
            hipMalloc(&d_leftStrings, leftSize * sizeof(char*));
            hipMalloc(&d_rightStrings, rightSize * sizeof(char*));
            hipMalloc(&d_results, resultSize * sizeof(uint8_t));
            
            hipMemcpy(d_leftStrings, h_leftStrings, leftSize * sizeof(char*), hipMemcpyHostToDevice);
            hipMemcpy(d_rightStrings, h_rightStrings, rightSize * sizeof(char*), hipMemcpyHostToDevice);
            
            // Launch string comparison kernel
            compareStringColumns<<<gridDim, blockDim>>>(
                d_leftStrings, d_rightStrings, leftSize, rightSize, d_results, opType);
            
            // Copy results back to host
            hipMemcpy(resultVector.data(), d_results, resultSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
            
            // Free device memory
            for (int i = 0; i < leftSize; i++) {
                hipFree(h_leftStrings[i]);
            }
            
            for (int i = 0; i < rightSize; i++) {
                hipFree(h_rightStrings[i]);
            }
            
            hipFree(d_leftStrings);
            hipFree(d_rightStrings);
            hipFree(d_results);
            
            delete[] h_leftStrings;
            delete[] h_rightStrings;
        }
    }
    
    return resultVector;
}

std::vector<uint8_t> GPUManager::processBinaryExpr(
    const Table& leftTable, 
    const Table& rightTable,
    const hsql::Expr* expr) 
{
    // This is a simplified implementation
    if (expr->type == hsql::kExprOperator) {
        if (expr->opType == hsql::OperatorType::kOpAnd || expr->opType == hsql::OperatorType::kOpOr) {
              // Process binary kOpAnd/kOpOr operations
              auto leftResults = processBinaryExpr(leftTable, rightTable, expr->expr);
              auto rightResults = processBinaryExpr(leftTable, rightTable, expr->expr2);
              
              int resultSize = leftTable.getSize() * rightTable.getSize();
              
              // Create device vectors
              uint8_t *d_leftResults, *d_rightResults, *d_output;
              hipMalloc(&d_leftResults, resultSize * sizeof(uint8_t));
              hipMalloc(&d_rightResults, resultSize * sizeof(uint8_t));
              hipMalloc(&d_output, resultSize * sizeof(uint8_t));
              
              // Copy data to device
              hipMemcpy(d_leftResults, leftResults.data(), resultSize * sizeof(uint8_t), hipMemcpyHostToDevice);
              hipMemcpy(d_rightResults, rightResults.data(), resultSize * sizeof(uint8_t), hipMemcpyHostToDevice);
              
              // Set up kernel execution parameters
              int blockSize = 256;
              int numBlocks = (resultSize + blockSize - 1) / blockSize;
              
              // Execute kernel
              uint8_t isAnd = expr->opType == hsql::OperatorType::kOpAnd ? 1 : 0;
              combineResults<<<numBlocks, blockSize>>>(d_leftResults, d_rightResults, d_output, resultSize, isAnd);
              
              // Copy results back to host
              std::vector<uint8_t> resultVector(resultSize);
              hipMemcpy(resultVector.data(), d_output, resultSize * sizeof(uint8_t), hipMemcpyDeviceToHost);
              
              // Free device memory
              hipFree(d_leftResults);
              hipFree(d_rightResults);
              hipFree(d_output);
              
              return resultVector;
        } else {
            return processComparisonExpr(leftTable, rightTable, expr);
        }
    }
    
    // Default case - no conditions
    int resultSize = leftTable.getSize() * rightTable.getSize();
    return std::vector<uint8_t>(resultSize, 1);  // 1 means true
}
